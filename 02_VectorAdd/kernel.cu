#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define THREADS_PER_BLOCK 16	// Threads per block
#define SIZE 65			// Array size

__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
	// blockIdx.x is block index
	// threadIdx.x is thread index
	// blockDim.x corresponds to threads per block

	int i = threadIdx.x + blockIdx.x * blockDim.x;

	// Avoid accessing beyond the end of the arrays
	if (i < n)
		c[i] = a[i] + b[i];

	// Parallel threads
	// c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];

	// Parallel blocks
	// c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main()
{
	int block_size = SIZE / THREADS_PER_BLOCK;
	int *a, *b, *c;		// Host arrays
	int *d_a, *d_b, *d_c;	// Device arrays

	// Allocate the memory on the CPU
	a = (int *)malloc(SIZE * sizeof(int));
	b = (int *)malloc(SIZE * sizeof(int));
	c = (int *)malloc(SIZE * sizeof(int));

	// Allocate the memory on the GPU
	hipMalloc(&d_a, SIZE * sizeof(int));
	hipMalloc(&d_b, SIZE * sizeof(int));
	hipMalloc(&d_c, SIZE * sizeof(int));

	for (int i = 0; i < SIZE; ++i)
	{
		a[i] = i;
		b[i] = i + 2;
	}

	// Copy Host array to Device array
	hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	// Make a call to GPU kernel
	vectorAdd <<< block_size, THREADS_PER_BLOCK >>>(d_a, d_b, d_c, SIZE);

	// Copy result back to Host array from Device array
	hipMemcpy(c, d_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] = %d\n", i, c[i]);

	// Free the Host array memory
	free(a);
	free(b);
	free(c);

	// Free the Device array memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
