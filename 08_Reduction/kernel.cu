﻿#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"


#include <stdio.h>

#include <stdio.h>
#include <time.h>
#include <ctime>

#define BLOCK_DIM 4
#define ARRAY_SIZE 12

__global__ void reduction(int *d_in, int *d_out)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;

	// Do reduction in global mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			d_in[i] += d_in[i + s];
		}
		__syncthreads(); // Make sure all adds at one stage are done!
	}

	if (tid == 0)
	{
		d_out[blockIdx.x] = d_in[i];
	}
}

int main()
{
	const int N = ARRAY_SIZE;
	srand(time(NULL));

	int d[ARRAY_SIZE + BLOCK_DIM], a[12] = { 1, 3, 21, 55, 2, 5, 6, 8, 87, 6, 5, 0 };

	int *dev_a, *dev_d;

	hipMalloc((void **)&dev_a, N * sizeof(int));
	hipMalloc((void **)&dev_d, BLOCK_DIM * sizeof(int));

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);

	reduction <<< (ARRAY_SIZE + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM >>>(dev_a, dev_d);

	hipDeviceSynchronize();

	hipMemcpy(d, dev_d, ARRAY_SIZE / BLOCK_DIM * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < ARRAY_SIZE / BLOCK_DIM; ++i)
		printf("d[%d]: %d\n", i, d[i]);

	hipFree(dev_a);
	hipFree(dev_d);

	printf("");

	return 0;

	return 0;
}