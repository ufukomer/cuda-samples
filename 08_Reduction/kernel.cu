#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void reduction(float *d_out, float *d_in)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int tid = threadIdx.x;

	// Do reduction in global mem
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			d_in[i] += d_in[i + s];
		}
		__syncthreads(); // Make sure all adds at one stage are done!
	}
	
	if (tid == 0)
	{
		d_out[blockIdx.x] = d_in[i];
	}
}

int main()
{
    // TODO: Write main function

    return 0;
}