#include "hip/hip_runtime.h"


#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	printf("%c\n", STR[threadIdx.x]);
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 1;

	hello <<< num_blocks, num_threads >>>();

	hipDeviceSynchronize();

	return 0;
}