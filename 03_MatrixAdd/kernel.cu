#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCK_DIM 16

// Column & Row size
const int N = 16;
const int SIZE = N * N;

__global__ void matrixAdd(int *c, const int *a, const int *b, int n)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int i = col + row * n;

	if (row < n && col < n)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	int a[N][N], b[N][N], *c;
	int *dev_a, *dev_b, *dev_c;

	c = (int *)malloc(SIZE * sizeof(int));

	hipMalloc((void **)&dev_a, SIZE * sizeof(int));
	hipMalloc((void **)&dev_b, SIZE * sizeof(int));
	hipMalloc((void **)&dev_c, SIZE * sizeof(int));

	for (int row = 0; row < N; ++row)
		for (int col = 0; col < N; ++col)
		{
			a[row][col] = col;
			b[row][col] = col + 2;
		}

	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCK_DIM, BLOCK_DIM);
	dim3 dimGrid((int)ceil(N / dimBlock.x), (int)ceil(N / dimBlock.y));

	matrixAdd <<< dimGrid, dimBlock >>>(dev_c, dev_a, dev_b, N);

	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] =  %d\n", i, c[i]);

	// Free the Host array memory
	free(c);

	// Free the Device array memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}