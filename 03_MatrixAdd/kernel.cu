#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h> 

// Row size
const int N = 3;
// Column Size
const int M = 4;
const int SIZE = M * N;

__global__ void matrixAdd(int *c, const int *a, const int *b, int cols, int rows)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	// 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11
	int i = row * cols + col;

	if (row < rows && col < cols)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	srand(time(NULL));

	int a[N][M], b[N][M], *c;
	int *dev_a, *dev_b, *dev_c;

	c = (int *)malloc(SIZE * sizeof(int));

	hipMalloc((void **)&dev_a, SIZE * sizeof(int));
	hipMalloc((void **)&dev_b, SIZE * sizeof(int));
	hipMalloc((void **)&dev_c, SIZE * sizeof(int));

	for (int row = 0; row < N; ++row)
		for (int col = 0; col < M; ++col)
		{
			a[row][col] = rand() % 10;
			b[row][col] = rand() % 10;
		}

	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(M, N);
	dim3 dimGrid((int)ceil(M / dimBlock.x), (int)ceil(N / dimBlock.y));

	matrixAdd <<< dimGrid, dimBlock >>>(dev_c, dev_a, dev_b, M, N);

	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] =  %d\n", i, c[i]);

	// Free the Host array memory
	free(c);

	// Free the Device array memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
