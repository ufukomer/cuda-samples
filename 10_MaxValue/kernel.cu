﻿#pragma once
#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#include "hip/hip_runtime.h"


#include <stdio.h>


#include <stdio.h>
#include <time.h>
#include <ctime>

#define BLOCK_DIM 4
#define ARRAY_SIZE 12

__global__ void maxValue(int *a, int *d)
{
	__shared__ int sdata[BLOCK_DIM]; 

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	sdata[tid] = a[i];

	for (unsigned int s = BLOCK_DIM / 2; s >= 1; s = s / 2)
	{
		if (tid < s)
		{
			if (sdata[tid] < sdata[tid + s])
			{
				sdata[tid] = sdata[tid + s];
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		d[blockIdx.x] = sdata[0];
	}
}

int main()
{
	srand(time(NULL));
	const int N = ARRAY_SIZE;

	int d[ARRAY_SIZE / BLOCK_DIM], a[12] = { 1, 3, 21, 55, 2, 5, 6, 8, 87, 6, 5, 0 };

	int *dev_a, *dev_d;

	hipMalloc((void **)&dev_a, N * sizeof(int));
	hipMalloc((void **)&dev_d, BLOCK_DIM * sizeof(int));

	hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);

	maxValue <<< (ARRAY_SIZE + BLOCK_DIM - 1) / BLOCK_DIM, BLOCK_DIM >>>(dev_a, dev_d);

	hipDeviceSynchronize();

	hipMemcpy(d, dev_d, ARRAY_SIZE / BLOCK_DIM * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < ARRAY_SIZE / BLOCK_DIM; ++i)
		printf("Max: %d\n", d[i]);

	hipFree(dev_a);
	hipFree(dev_d);

	return 0;
}