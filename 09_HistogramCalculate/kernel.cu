﻿#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h> 
#include <iostream>

#define ARRAY_SIZE 1024
#define BLOCK_DIM 1024

using namespace std;

__global__ void fill_histrogram(int *dev_out, int *dev_in)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	atomicAdd(&dev_out[dev_in[i]], 1);
}

int main()
{
	int a[ARRAY_SIZE], b[100];
	int *dev_in, *dev_out;

	srand(time(NULL));

	hipMalloc((void **)&dev_in, ARRAY_SIZE * sizeof(int));
	hipMalloc((void **)&dev_out, 100 * sizeof(int));

	for (int i = 0; i < ARRAY_SIZE; ++i)
	{
		a[i] = rand() % 100;
	}

	hipMemcpy(dev_in, a, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

	fill_histrogram <<< ARRAY_SIZE / BLOCK_DIM, BLOCK_DIM >>>(dev_out, dev_in);

	hipMemcpy(b, dev_out, 100 * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < 100; ++i)
	{
		cout << "Out[" << i << "]: " << b[i] << endl;
	}

	hipFree(dev_in);
	hipFree(dev_out);

	return 0;
}