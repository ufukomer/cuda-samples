#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h> 

#define BLOCK_DIM 16

// Column & Row size
const int N = 2;
const int SIZE = N * N;

__global__ void matrixMult(int *c, int *a, int *b, int n)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	int k, sum = 0;

	if (row < n && col < n)
	{
		for (k = 0; k < n; k++)
			sum += a[row * n + k] * b[k * n + col];

		c[row * n + col] = sum;
	}
}

int main()
{
	srand(time(NULL));

	int a[N][N], b[N][N], *c;
	int *dev_a, *dev_b, *dev_c;

	c = (int *)malloc(SIZE * sizeof(int));

	hipMalloc((void **)&dev_a, SIZE * sizeof(int));
	hipMalloc((void **)&dev_b, SIZE * sizeof(int));
	hipMalloc((void **)&dev_c, SIZE * sizeof(int));

	for (int row = 0; row < N; ++row)
		for (int col = 0; col < N; ++col)
		{
			a[row][col] = rand() % 10;
			b[row][col] = rand() % 10;
		}

	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(1, 1);
	dim3 dimGrid(N, N);

	matrixMult <<< dimGrid, dimBlock >>>(dev_c, dev_a, dev_b, N);

	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] =  %d\n", i, c[i]);

	// Free the Host array memory
	free(c);

	// Free the Device array memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}