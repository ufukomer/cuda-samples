#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h> 

#define BLOCK_DIM 16

// Row Size & Column Size
const int N = 2;
const int SIZE = N * N;

__global__ void matrixMult(int *c, int *a, int *b, int n)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	int sum = 0;

	if (row < n && col < n)
	{
		for (int i = 0; i < n; ++i)
			sum += a[row * n + i] * b[i * n + col];

		c[row * n + col] = sum;
	}
}

int main()
{
	srand(time(NULL));

	int a[N][N] = { { 1, 2 },
			{ 2, 1 } };

	int b[N][N] = { { 1, 2 },
			{ 2, 1 } };

	int *c;
	int *dev_a, *dev_b, *dev_c;

	c = (int *)malloc(SIZE * sizeof(int));

	hipMalloc((void **)&dev_a, SIZE * sizeof(int));
	hipMalloc((void **)&dev_b, SIZE * sizeof(int));
	hipMalloc((void **)&dev_c, SIZE * sizeof(int));

	hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimBlock(N, N);
	dim3 dimGrid((int)ceil(N / dimBlock.x), (int)ceil(N / dimBlock.y));

	matrixMult <<< dimGrid, dimBlock >>>(dev_c, dev_a, dev_b, N);

	hipMemcpy(c, dev_c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < SIZE; ++i)
		printf("c[%d] =  %d\n", i, c[i]);

	// Free the Host array memory
	free(c);

	// Free the Device array memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
